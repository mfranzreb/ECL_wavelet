#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>
#include <nvbio/basic/packed_vector.h>
#include <nvbio/strings/alphabet.h>
#include <nvbio/strings/wavelet_tree.h>

#include <random>
#include <utils.cuh>
#include <wavelet_tree.cuh>

#include "sdsl/wavelet_trees.hpp"
#include "test_benchmark_utils.cuh"

namespace ecl {

template <typename T>
static void BM_Access(benchmark::State& state) {
  auto const data_size = state.range(0);
  auto const alphabet_size = state.range(1);
  auto const num_queries = state.range(2);

  auto alphabet = std::vector<T>(alphabet_size);
  std::iota(alphabet.begin(), alphabet.end(), 0);
  auto data = generateRandomData<T>(alphabet, data_size);

  state.counters["param.data_size"] = data_size;
  state.counters["param.alphabet_size"] = alphabet_size;
  state.counters["param.num_queries"] = num_queries;

  auto queries = generateRandomQueries(data_size, num_queries);

  auto wt = WaveletTree<T>(data.data(), data_size, std::move(alphabet), 0);

  for (auto _ : state) {
    auto results = wt.template access<1>(queries.data(), num_queries);
  }
}

template <nvbio::Alphabet AlphabetType>
nvbio::PackedVector<nvbio::device_tag,
                    nvbio::AlphabetTraits<AlphabetType>::SYMBOL_SIZE, true>
getNVbioArgs(size_t const data_size, uint8_t const alphabet_size = 0) {
  std::vector<uint8_t> alphabet;
  std::vector<uint8_t> data(data_size);
  if constexpr (AlphabetType == nvbio::DNA) {
    alphabet = std::vector<uint8_t>{'A', 'C', 'G', 'T'};
    data = generateRandomData<uint8_t>(alphabet, data_size);
  } else if constexpr (AlphabetType == nvbio::DNA_N) {
    alphabet = std::vector<uint8_t>{'A', 'C', 'G', 'T', 'N'};
    data = generateRandomData<uint8_t>(alphabet, data_size);
  } else if constexpr (AlphabetType == nvbio::PROTEIN) {
    alphabet = std::vector<uint8_t>{'A', 'C', 'D', 'E', 'F', 'G', 'H', 'I',
                                    'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'R',
                                    'S', 'T', 'V', 'W', 'Y', 'B', 'Z', 'X'};
    data = generateRandomData<uint8_t>(alphabet, data_size);
  } else if constexpr (AlphabetType == nvbio::ASCII) {
    std::tie(alphabet, data) =
        generateRandomAlphabetAndData<uint8_t>(alphabet_size, data_size, true);
  }
  uint32_t const alphabet_bits =
      nvbio::AlphabetTraits<AlphabetType>::SYMBOL_SIZE;

  // allocate a host packed vector
  nvbio::PackedVector<nvbio::host_tag, alphabet_bits, true> h_data(data_size);

  // pack the string
  nvbio::from_string<AlphabetType>(
      reinterpret_cast<const char*>(data.data()),
      reinterpret_cast<const char*>(data.data() + data.size()), h_data.begin());

  // copy it to the device
  nvbio::PackedVector<nvbio::device_tag, alphabet_bits, true> d_data(h_data);

  return d_data;
}

static void BM_NVBIO(benchmark::State& state) {
  if (state.range(0) > std::numeric_limits<uint32_t>::max()) {
    state.SkipWithError("Data size is too large for NVBIO.");
    return;
  }
  uint32_t const data_size = static_cast<uint32_t>(state.range(0));
  uint8_t const alphabet_size = static_cast<uint8_t>(state.range(1));
  auto const num_queries = state.range(2);

  state.counters["param.data_size"] = data_size;
  state.counters["param.alphabet_size"] = alphabet_size;

  auto queries = generateRandomQueries(data_size, num_queries);

  if (alphabet_size == 4) {
    auto d_data = getNVbioArgs<nvbio::DNA>(data_size);
    nvbio::WaveletTreeStorage<nvbio::device_tag> wt;
    nvbio::setup(data_size, d_data.begin(), wt);
    auto const wt_view = nvbio::plain_view(
        (const nvbio::WaveletTreeStorage<nvbio::device_tag>&)wt);
    for (auto _ : state) {
#pragma omp parallel for
      for (auto const& query : queries) {
        nvbio::text(wt_view, query);
      }
    }
  } else if (alphabet_size == 5) {
    auto d_data = getNVbioArgs<nvbio::DNA_N>(data_size);
    nvbio::WaveletTreeStorage<nvbio::device_tag> wt;
    nvbio::setup(data_size, d_data.begin(), wt);
    auto const wt_view = nvbio::plain_view(
        (const nvbio::WaveletTreeStorage<nvbio::device_tag>&)wt);
    for (auto _ : state) {
#pragma omp parallel for
      for (auto const& query : queries) {
        nvbio::text(wt_view, query);
      }
    }
  } else if (alphabet_size == 24) {
    auto d_data = getNVbioArgs<nvbio::PROTEIN>(data_size);
    nvbio::WaveletTreeStorage<nvbio::device_tag> wt;
    nvbio::setup(data_size, d_data.begin(), wt);
    auto const wt_view = nvbio::plain_view(
        (const nvbio::WaveletTreeStorage<nvbio::device_tag>&)wt);
    for (auto _ : state) {
#pragma omp parallel for
      for (auto const& query : queries) {
        nvbio::text(wt_view, query);
      }
    }
  } else {
    auto d_data = getNVbioArgs<nvbio::ASCII>(data_size, alphabet_size);
    nvbio::WaveletTreeStorage<nvbio::device_tag> wt;
    nvbio::setup(data_size, d_data.begin(), wt);
    auto const wt_view = nvbio::plain_view(
        (const nvbio::WaveletTreeStorage<nvbio::device_tag>&)wt);
    for (auto _ : state) {
#pragma omp parallel for
      for (auto const& query : queries) {
        nvbio::text(wt_view, query);
      }
    }
  }
}

// SDSL sometimes returns wrong result, "construct_im" ffunction does not work
// properly.
template <typename T>
static void BM_SDSL(benchmark::State& state) {
  auto const data_size = state.range(0);
  auto const alphabet_size = state.range(1);
  auto const num_queries = state.range(2);

  state.counters["param.data_size"] = data_size;
  state.counters["param.alphabet_size"] = alphabet_size;
  state.counters["param.num_queries"] = num_queries;

  auto queries = generateRandomQueries(data_size, num_queries);

  auto [alphabet, data] =
      generateRandomAlphabetAndData<T>(alphabet_size, data_size, true);

  // write data to file
  std::ofstream data_file("data_file");
  data_file.write(reinterpret_cast<const char*>(data.data()),
                  data.size() * sizeof(T));
  data_file.close();

  std::vector<T> results_sdsl(num_queries);
  if constexpr (sizeof(T) == 1) {
    sdsl::wt_pc<sdsl::balanced_shape, sdsl::bit_vector, sdsl::rank_support_v5<>>
        wt;
    sdsl::construct(wt, "data_file", sizeof(T));

    // delete file
    std::remove("data_file");

    for (auto _ : state) {
#pragma omp parallel for
      for (size_t i = 0; i < num_queries; ++i) {
        results_sdsl[i] = wt[queries[i]];
      }
    }
  } else {
    sdsl::wt_pc<sdsl::balanced_shape, sdsl::bit_vector, sdsl::rank_support_v5<>,
                sdsl::wt_pc<sdsl::balanced_shape>::select_1_type,
                sdsl::wt_pc<sdsl::balanced_shape>::select_0_type,
                sdsl::int_tree<>>
        wt;
    sdsl::construct(wt, "data_file", sizeof(T));

    // delete file
    std::remove("data_file");

    for (auto _ : state) {
#pragma omp parallel for
      for (size_t i = 0; i < num_queries; ++i) {
        results_sdsl[i] = wt[queries[i]];
      }
    }
  }
}

// For initializing CUDA
BENCHMARK(BM_Access<uint8_t>)
    ->Args({100, 4, 1})
    ->Iterations(10)
    ->Unit(benchmark::kMillisecond);

// First argument is the size of the data, second argument is the size of the
// alphabet.
BENCHMARK(BM_Access<uint8_t>)
    ->ArgsProduct({{200'000'000, 500'000'000, 800'000'000, 1'000'000'000,
                    1'500'000'000, 2'000'000'000},
                   {4, 5, 24, 64, 100, 155, 250},
                   {100, 1'000, 5'000, 10'000, 50'000, 100'000}})
    ->Iterations(5)
    ->Unit(benchmark::kMillisecond);

// BENCHMARK(BM_Access<uint16_t>)
//     ->ArgsProduct({{200'000'000, 500'000'000, 800'000'000, 1'000'000'000,
//                     1'200'000'000},
//                    {500, 1'000, 2'000, 4'000, 8'000, 16'000, 32'000,
//                    64'000},
//{100, 1'000, 5'000, 10'000, 50'000, 100'000}})
//     ->Iterations(5)
//     ->Unit(benchmark::kMillisecond);

BENCHMARK(BM_NVBIO)
    ->ArgsProduct({{200'000'000, 500'000'000, 800'000'000, 1'000'000'000,
                    1'500'000'000, 2'000'000'000},
                   {4, 5, 24, 64, 100, 155, 250},
                   {100, 1'000, 5'000, 10'000, 50'000, 100'000}})
    ->Iterations(5)
    ->Unit(benchmark::kMillisecond);

BENCHMARK(BM_SDSL<uint8_t>)
    ->ArgsProduct({{200'000'000, 500'000'000, 800'000'000, 1'000'000'000,
                    1'500'000'000, 2'000'000'000},
                   {4, 5, 24, 64, 100, 155, 250},
                   {100, 1'000, 5'000, 10'000, 50'000, 100'000}})
    ->Iterations(5)
    ->Unit(benchmark::kMillisecond);

// BENCHMARK(BM_SDSL<uint16_t>)
//     ->ArgsProduct({{200'000'000, 500'000'000, 800'000'000, 1'000'000'000,
//                     1'200'000'000},
//                    {500, 1'000, 2'000, 4'000, 8'000, 16'000, 32'000,
//                    64'000},
//{100, 1'000, 5'000, 10'000, 50'000, 100'000}})
//     ->Iterations(5)
//     ->Unit(benchmark::kMillisecond);

}  // namespace ecl