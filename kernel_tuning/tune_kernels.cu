#include "hip/hip_runtime.h"
#include <algorithm>
#include <bit_array.cuh>
#include <chrono>
#include <cstdint>
#include <rank_select.cuh>
#include <test_benchmark_utils.cuh>
#include <utils.cuh>
#include <vector>
#include <wavelet_tree.cuh>

namespace ecl {

void tuneQueries(std::string out_dir, uint32_t const GPU_index) {
  uint8_t const num_iters = 1;
  auto const prop = getDeviceProperties();
  struct hipFuncAttributes funcAttrib;
  gpuErrchk(hipFuncGetAttributes(&funcAttrib,
                                  reinterpret_cast<const void*>(accessKernel<uint8_t), true, 1, true, true>));
  uint32_t max_size_access =
      std::min(kMaxTPB, static_cast<uint32_t>(funcAttrib.maxThreadsPerBlock));
  gpuErrchk(hipFuncGetAttributes(&funcAttrib,
                                  reinterpret_cast<const void*>(rankKernel<uint8_t), true, 1, true, true>));
  uint32_t max_size_rank =
      std::min(kMaxTPB, static_cast<uint32_t>(funcAttrib.maxThreadsPerBlock));
  gpuErrchk(hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(selectKernel<uint8_t), 1, true>));
  uint32_t max_size_select =
      std::min(kMaxTPB, static_cast<uint32_t>(funcAttrib.maxThreadsPerBlock));
  size_t const data_size = prop.totalGlobalMem / 10;

  auto const GPU_name = prop.name;

  // Tune chunks vs access_queries
  std::string out_file_access = out_dir + "/access_chunks_vs_queries.csv";
  std::string out_file_rank = out_dir + "/rank_chunks_vs_queries.csv";
  std::string out_file_select = out_dir + "/select_chunks_vs_queries.csv";
  // Write column names to CSV
  std::ofstream file(out_file_access);
  file << "num_chunks,num_queries,time,GPU_name" << std::endl;
  file.close();
  file = std::ofstream(out_file_rank);
  file << "num_chunks,num_queries,time,GPU_name" << std::endl;
  file.close();
  file = std::ofstream(out_file_select);
  file << "num_chunks,num_queries,time,GPU_name" << std::endl;
  file.close();

  std::vector<uint8_t> num_chunks_vec({2, 4, 6, 8, 10, 12, 14, 16, 18, 20});
  std::vector<uint32_t> num_queries_vec({100'000, 500'000, 1'000'000, 5'000'000,
                                         10'000'000, 50'000'000, 100'000'000});
  size_t const alphabet_size = 16;

  std::vector<uint8_t> alphabet(alphabet_size);
  std::iota(alphabet.begin(), alphabet.end(), 0);
  auto [data, hist] = generateRandomDataAndHist(alphabet, data_size);
  auto alphabet_copy = alphabet;

  WaveletTree<uint8_t> wt(data.data(), data_size, std::move(alphabet_copy),
                          GPU_index);

  // create graphs for each number of chunks
  for (auto chunk : num_chunks_vec) {
    queries_graph_cache[chunk] = createQueriesGraph(chunk, 2);
  }

  std::chrono::high_resolution_clock::time_point start_time, end_time;
  IdealConfigs& ideal_configs = getIdealConfigs(GPU_name);
  for (uint32_t num_query : num_queries_vec) {
    auto access_queries =
        generateRandomAccessQueries(data_size, static_cast<size_t>(num_query));
    auto rank_queries = generateRandomRankQueries(
        data_size, static_cast<size_t>(num_query), alphabet);
    auto select_queries = generateRandomSelectQueries(
        hist, static_cast<size_t>(num_query), alphabet);
    gpuErrchk(hipHostRegister(access_queries.data(),
                               num_query * sizeof(size_t),
                               hipHostRegisterDefault));
    gpuErrchk(hipHostRegister(rank_queries.data(),
                               num_query * sizeof(RankSelectQuery<uint8_t>),
                               hipHostRegisterDefault));
    gpuErrchk(hipHostRegister(select_queries.data(),
                               num_query * sizeof(RankSelectQuery<uint8_t>),
                               hipHostRegisterDefault));

    for (auto num_chunk : num_chunks_vec) {
      // Set ideal_configs slope so that correct num_chunks is chosen
      float const slope = static_cast<float>(num_chunk) /
                          std::log(static_cast<float>(num_query));
      ideal_configs.accessKernel_logrel.slope = slope;
      ideal_configs.rankKernel_logrel.slope = slope;
      ideal_configs.selectKernel_logrel.slope = slope;
      // Warmup
      for (uint8_t i = 0; i < 2; ++i) {
        auto results = wt.template access<1>(access_queries.data(), num_query);
      }

      std::vector<size_t> times(num_iters);
      for (uint8_t i = 0; i < num_iters; ++i) {
        start_time = std::chrono::high_resolution_clock::now();
        auto results = wt.template access<1>(access_queries.data(), num_query);
        end_time = std::chrono::high_resolution_clock::now();
        times[i] = std::chrono::duration_cast<std::chrono::microseconds>(
                       end_time - start_time)
                       .count();
      }
      // Write median time to CSV
      std::nth_element(times.begin(), times.begin() + times.size() / 2,
                       times.end());
      std::ofstream file(out_file_access, std::ios_base::app);
      file << +num_chunk << "," << num_query << "," << times[num_iters / 2]
           << "," << GPU_name << std::endl;
      file.close();

      // Warmup
      for (uint8_t i = 0; i < 2; ++i) {
        auto results = wt.template rank<1>(rank_queries.data(), num_query);
      }

      for (uint8_t i = 0; i < num_iters; ++i) {
        start_time = std::chrono::high_resolution_clock::now();
        auto results = wt.template rank<1>(rank_queries.data(), num_query);
        end_time = std::chrono::high_resolution_clock::now();
        times[i] = std::chrono::duration_cast<std::chrono::microseconds>(
                       end_time - start_time)
                       .count();
      }
      // Write median time to CSV
      std::nth_element(times.begin(), times.begin() + times.size() / 2,
                       times.end());
      file = std::ofstream(out_file_rank, std::ios_base::app);
      file << +num_chunk << "," << num_query << "," << times[num_iters / 2]
           << "," << GPU_name << std::endl;
      file.close();

      // Warmup
      for (uint8_t i = 0; i < 2; ++i) {
        auto results = wt.template select<1>(select_queries.data(), num_query);
      }

      for (uint8_t i = 0; i < num_iters; ++i) {
        start_time = std::chrono::high_resolution_clock::now();
        auto results = wt.template select<1>(select_queries.data(), num_query);
        end_time = std::chrono::high_resolution_clock::now();
        times[i] = std::chrono::duration_cast<std::chrono::microseconds>(
                       end_time - start_time)
                       .count();
      }
      // Write median time to CSV
      std::nth_element(times.begin(), times.begin() + times.size() / 2,
                       times.end());
      file = std::ofstream(out_file_select, std::ios_base::app);
      file << +num_chunk << "," << num_query << "," << times[num_iters / 2]
           << "," << GPU_name << std::endl;
      file.close();
    }
    gpuErrchk(hipHostUnregister(access_queries.data()));
    gpuErrchk(hipHostUnregister(rank_queries.data()));
    gpuErrchk(hipHostUnregister(select_queries.data()));
  }
}
}  // namespace ecl

int main(int argc, char* argv[]) {
  auto const parent_dir = argv[1];
  auto const GPU_index = std::stoi(argv[2]);
  ecl::checkWarpSize(GPU_index);
  ecl::tuneQueries(std::string(parent_dir), GPU_index);
  return 0;
}