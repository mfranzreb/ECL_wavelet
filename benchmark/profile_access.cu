#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include <random>
#include <wavelet_tree.cuh>

#include "test_benchmark_utils.cuh"
#include "utils.cuh"

template <typename T>
void profileAccess(size_t const data_size, size_t const alphabet_size,
                   size_t const num_queries, bool const use_profiler_api) {
  std::vector<T> alphabet;
  std::vector<T> data;
  alphabet = std::vector<T>(alphabet_size);
  std::iota(alphabet.begin(), alphabet.end(), 0);
  data = ecl::generateRandomData<T>(alphabet, data_size);
  auto queries = ecl::generateRandomAccessQueries(data_size, num_queries);
  ecl::WaveletTree<T> wt(data.data(), data_size, std::move(alphabet), 0);
  auto results = wt.access(queries.data(), num_queries);
  if (use_profiler_api) {
    hipProfilerStart();
    results = wt.access(queries.data(), num_queries);
    hipProfilerStop();
  }
}

int main(int argc, char** argv) {
  // size is first command line argument
  auto const data_size = std::stoul(argv[1]);
  auto const alphabet_size = std::stoul(argv[2]);
  auto const num_queries = std::stoul(argv[3]);
  bool const use_profiler_api = argc > 4 ? std::stoi(argv[4]) : false;

  auto queries = ecl::generateRandomAccessQueries(data_size, num_queries);

  if (alphabet_size < std::numeric_limits<uint8_t>::max()) {
    profileAccess<uint8_t>(data_size, alphabet_size, num_queries,
                           use_profiler_api);
  } else if (alphabet_size < std::numeric_limits<uint16_t>::max()) {
    profileAccess<uint16_t>(data_size, alphabet_size, num_queries,
                            use_profiler_api);
  }
  return 0;
}