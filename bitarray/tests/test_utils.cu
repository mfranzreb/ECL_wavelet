#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include "utils.cuh"

namespace ecl {

template <typename T>
class UtilsTest : public ::testing::Test {
 protected:
  T *result;
  void SetUp() override { gpuErrchk(hipMallocManaged(&result, sizeof(T))); }
  void TearDown() override { gpuErrchk(hipFree(result)); }
};

template <typename T>
__global__ void getPrevPowTwoKernel(T n, T *result) {
  *result = getPrevPowTwo(n);
}

using MyTypes = testing::Types<uint8_t, uint16_t, uint32_t, uint64_t>;
TYPED_TEST_SUITE(UtilsTest, MyTypes);

TYPED_TEST(UtilsTest, getPrevPowTwo) {
  TypeParam limit = std::min(
      size_t(1024), static_cast<size_t>(std::numeric_limits<TypeParam>::max()));
  TypeParam expected = 0;
  for (TypeParam i = 0; i < limit; ++i) {
    getPrevPowTwoKernel<TypeParam><<<1, 1>>>(i, this->result);
    kernelCheck();
    EXPECT_EQ(*this->result, expected);
    if (isPowTwo(i)) {
      expected = i;
    }
  }
}
}  // namespace ecl