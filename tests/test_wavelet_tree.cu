#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <algorithm>
#include <fstream>
#include <random>
#include <unordered_map>
#include <vector>
#include <wavelet_tree.cuh>

#include "test_benchmark_utils.cuh"

namespace ecl {

static constexpr uint8_t kGPUIndex = 0;

template <typename T>
class WaveletTreeTest : public WaveletTree<T> {
 public:
  using WaveletTree<T>::WaveletTree;
  using WaveletTree<T>::computeGlobalHistogram;
};

template <typename T>
class WaveletTreeTestFixture : public ::testing::Test {
 protected:
  T* result;
  void SetUp() override { gpuErrchk(hipMallocManaged(&result, sizeof(T))); }
  void TearDown() override { gpuErrchk(hipFree(result)); }
};

template <typename T>
std::pair<std::vector<T>, std::vector<T>> generateRandomAlphabetDataAndHist(
    size_t alphabet_size, size_t const data_size,
    std::unordered_map<T, size_t>& hist) {
  std::vector<T> alphabet(alphabet_size);
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<T> dis(0, std::numeric_limits<T>::max());
  std::generate(alphabet.begin(), alphabet.end(), [&]() { return dis(gen); });
  // Check that all elements are unique
  std::sort(alphabet.begin(), alphabet.end());
  // remove duplicates
  auto it = std::unique(alphabet.begin(), alphabet.end());
  alphabet_size = std::distance(alphabet.begin(), it);
  alphabet.resize(alphabet_size);

  std::vector<T> data(data_size);
  std::uniform_int_distribution<size_t> dis2(0, alphabet_size - 1);
  std::generate(data.begin(), data.end(), [&]() {
    auto const symbol = alphabet[dis2(gen)];
    hist[symbol]++;
    return symbol;
  });

  return std::make_pair(alphabet, data);
}

template <typename T>
std::vector<size_t> calculateHistogram(const std::vector<T>& data,
                                       const std::vector<T>& alphabet) {
  std::vector<size_t> histogram(alphabet.size(), 0);

  // Count occurrences of each value in the data
  for (T const& value : data) {
    auto it = std::find(alphabet.begin(), alphabet.end(), value);
    if (it != alphabet.end()) {
      histogram[std::distance(alphabet.begin(), it)]++;
    }
  }
  return histogram;
}

template <typename T>
__global__ void BAaccessKernel(BitArray bit_array, size_t array_index,
                               size_t index, T* output) {
  *output = static_cast<T>(bit_array.access(array_index, index));
}

template <typename T, int NumThreads>
__host__ void compareAccessResults(WaveletTree<T>& wt,
                                   std::vector<size_t>& indices,
                                   std::vector<T> const& data) {
  auto const results = wt.template access<NumThreads>(indices);
  for (size_t i = 0; i < indices.size(); ++i) {
    EXPECT_EQ(data[indices[i]], results[i]);
  }
}

using MyTypes = testing::Types<uint8_t, uint16_t, uint32_t, uint64_t>;
TYPED_TEST_SUITE(WaveletTreeTestFixture, MyTypes);

TYPED_TEST(WaveletTreeTestFixture, WaveletTreeConstructor) {
  std::vector<TypeParam> data{1, 2, 3, 4, 5, 6, 7, 8, 9};
  {
    std::vector<TypeParam> alphabet{1, 2, 3, 4, 5, 6, 7, 8, 9};
    WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                              kGPUIndex);
  }
  {
    data = std::vector<TypeParam>{0, 1, 2, 3};
    std::vector<TypeParam> alphabet{0, 1, 2, 3};

    WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                              kGPUIndex);
  }
}

TYPED_TEST(WaveletTreeTestFixture, createMinimalCodes) {
  // Check that for powers of two, the vector is empty
  for (size_t i = 4; i < 8 * sizeof(TypeParam); i *= 2) {
    std::vector<TypeParam> alphabet(i);
    std::iota(alphabet.begin(), alphabet.end(), 0);
    auto codes = WaveletTree<TypeParam>::createMinimalCodes(alphabet);
    EXPECT_TRUE(codes.empty());
  }

  size_t alphabet_size = 72;
  std::vector<TypeParam> alphabet(alphabet_size);
  std::iota(alphabet.begin(), alphabet.end(), 0);
  auto codes = WaveletTree<TypeParam>::createMinimalCodes(alphabet);
  auto code_value = 64UL;
  for (int i = 64; i <= 71; ++i) {
    EXPECT_EQ(codes[i - 64].code_, code_value);
    EXPECT_EQ(codes[i - 64].len_, 4);
    code_value += 8;
  }

  alphabet_size = 63;
  alphabet.resize(alphabet_size);
  std::iota(alphabet.begin(), alphabet.end(), 0);
  codes = WaveletTree<TypeParam>::createMinimalCodes(alphabet);
  EXPECT_EQ(codes[0].code_, 62);
  EXPECT_EQ(codes[0].len_, 5);

  alphabet_size = 75;
  alphabet.resize(alphabet_size);
  std::iota(alphabet.begin(), alphabet.end(), 0);
  codes = WaveletTree<TypeParam>::createMinimalCodes(alphabet);
  code_value = 64UL;
  for (int i = 64; i < 72; ++i) {
    EXPECT_EQ(codes[i - 64].code_, code_value);
    EXPECT_EQ(codes[i - 64].len_, 5);
    code_value += 4;
  }
  EXPECT_EQ(codes[72 - 64].code_, 96);
  EXPECT_EQ(codes[72 - 64].len_, 4);
  EXPECT_EQ(codes[73 - 64].code_, 104);
  EXPECT_EQ(codes[73 - 64].len_, 4);
  EXPECT_EQ(codes[74 - 64].code_, 112);
  EXPECT_EQ(codes[74 - 64].len_, 3);
}

TYPED_TEST(WaveletTreeTestFixture, TestGlobalHistogram) {
  std::vector<TypeParam> alphabet{0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  auto const alphabet_size = alphabet.size();

  std::vector<TypeParam> data(1000);
  for (size_t i = 0; i < data.size(); ++i) {
    data[i] = i % alphabet_size;
  }

  // allocate memory for arguments of kernel
  TypeParam* d_alphabet;
  TypeParam* d_data;
  size_t* d_histogram;
  gpuErrchk(hipMalloc(&d_alphabet, sizeof(TypeParam) * alphabet_size));
  gpuErrchk(hipMemcpy(d_alphabet, alphabet.data(),
                       sizeof(TypeParam) * alphabet_size,
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMalloc(&d_data, sizeof(TypeParam) * data.size()));
  gpuErrchk(hipMemcpy(d_data, data.data(), sizeof(TypeParam) * data.size(),
                       hipMemcpyHostToDevice));
  gpuErrchk(hipMalloc(&d_histogram, sizeof(size_t) * alphabet_size));
  gpuErrchk(hipMemset(d_histogram, 0, sizeof(size_t) * alphabet_size));

  auto alphabet_copy = alphabet;
  // Create the wavelet tree
  WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet_copy),
                            kGPUIndex);
  computeGlobalHistogramKernel<TypeParam, true, false, false><<<1, 32>>>(
      wt, d_data, data.size(), d_histogram, d_alphabet, alphabet_size, 0);
  kernelCheck();

  // Pass the histogram to the host
  std::vector<size_t> h_histogram(alphabet_size);
  gpuErrchk(hipMemcpy(h_histogram.data(), d_histogram,
                       sizeof(size_t) * alphabet_size, hipMemcpyDeviceToHost));

  auto hist_should = calculateHistogram(data, alphabet);
  for (size_t i = 0; i < alphabet_size; ++i) {
    EXPECT_EQ(hist_should[i], h_histogram[i]);
  }

  data = std::vector<TypeParam>(1000, 0);
  gpuErrchk(hipMemcpy(d_data, data.data(), sizeof(TypeParam) * data.size(),
                       hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(d_histogram, 0, sizeof(size_t) * alphabet_size));
  computeGlobalHistogramKernel<TypeParam, true, false, false><<<1, 32>>>(
      wt, d_data, data.size(), d_histogram, d_alphabet, alphabet_size, 0);
  kernelCheck();

  gpuErrchk(hipMemcpy(h_histogram.data(), d_histogram,
                       sizeof(size_t) * alphabet_size, hipMemcpyDeviceToHost));

  hist_should = calculateHistogram(data, alphabet);
  for (size_t i = 0; i < alphabet_size; ++i) {
    EXPECT_EQ(hist_should[i], h_histogram[i]);
  }

  for (size_t i = 0; i < data.size(); ++i) {
    if (i < alphabet_size) {
      data[i] = i;
    } else {
      data[i] = alphabet_size - 1;
    }
  }

  gpuErrchk(hipMemcpy(d_data, data.data(), sizeof(TypeParam) * data.size(),
                       hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(d_histogram, 0, sizeof(size_t) * alphabet_size));
  computeGlobalHistogramKernel<TypeParam, true, false, false><<<1, 32>>>(
      wt, d_data, data.size(), d_histogram, d_alphabet, alphabet_size, 0);
  kernelCheck();

  gpuErrchk(hipMemcpy(h_histogram.data(), d_histogram,
                       sizeof(size_t) * alphabet_size, hipMemcpyDeviceToHost));

  hist_should = calculateHistogram(data, alphabet);
  for (size_t i = 0; i < alphabet_size; ++i) {
    EXPECT_EQ(hist_should[i], h_histogram[i]);
  }

  // Case where last two symbols have different counts
  data = std::vector<TypeParam>(1000, 0);
  for (size_t i = 0; i < 5; ++i) {
    data[i] = 9;
  }
  for (size_t i = 5; i < 15; ++i) {
    data[i] = 8;
  }
  for (size_t i = 15; i < data.size(); ++i) {
    data[i] = i % 8;
  }

  gpuErrchk(hipMemcpy(d_data, data.data(), sizeof(TypeParam) * data.size(),
                       hipMemcpyHostToDevice));

  gpuErrchk(hipMemset(d_histogram, 0, sizeof(size_t) * alphabet_size));
  computeGlobalHistogramKernel<TypeParam, true, false, false><<<1, 32>>>(
      wt, d_data, data.size(), d_histogram, d_alphabet, alphabet_size, 0);
  kernelCheck();

  gpuErrchk(hipMemcpy(h_histogram.data(), d_histogram,
                       sizeof(size_t) * alphabet_size, hipMemcpyDeviceToHost));

  hist_should = calculateHistogram(data, alphabet);
  for (size_t i = 0; i < alphabet_size; ++i) {
    EXPECT_EQ(hist_should[i], h_histogram[i]);
  }

  // Free memory
  gpuErrchk(hipFree(d_alphabet));
  gpuErrchk(hipFree(d_data));
  gpuErrchk(hipFree(d_histogram));
}

TYPED_TEST(WaveletTreeTestFixture, TestGlobalHistogramRandom) {
  size_t const data_size = 10000;
  TypeParam* d_data;
  gpuErrchk(hipMalloc(&d_data, sizeof(TypeParam) * data_size));
  for (int i = 0; i < 100; i++) {
    // Random alphabet size between 3 and data_size
    size_t alphabet_size =
        3 +
        (rand() %
         (std::min(static_cast<size_t>(std::numeric_limits<TypeParam>::max()),
                   data_size) -
          3));

    std::unordered_map<TypeParam, size_t> hist_should;
    auto [alphabet, data] = generateRandomAlphabetDataAndHist<TypeParam>(
        alphabet_size, data_size, hist_should);

    alphabet_size = alphabet.size();

    // allocate memory for arguments of kernel
    TypeParam* d_alphabet;
    size_t* d_histogram;
    gpuErrchk(hipMalloc(&d_alphabet, sizeof(TypeParam) * alphabet_size));
    gpuErrchk(hipMemcpy(d_alphabet, alphabet.data(),
                         sizeof(TypeParam) * alphabet_size,
                         hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&d_histogram, sizeof(size_t) * alphabet_size));
    gpuErrchk(hipMemset(d_histogram, 0, sizeof(size_t) * alphabet_size));

    gpuErrchk(hipMemcpy(d_data, data.data(), sizeof(TypeParam) * data_size,
                         hipMemcpyHostToDevice));

    // Create the wavelet tree
    auto alphabet_copy = alphabet;
    WaveletTreeTest<TypeParam> wt(data.data(), data_size,
                                  std::move(alphabet_copy), kGPUIndex);

    wt.computeGlobalHistogram(isPowTwo<TypeParam>(alphabet_size), data_size,
                              d_data, d_alphabet, d_histogram);

    // Pass the histogram to the host
    std::vector<size_t> h_histogram(alphabet_size);
    gpuErrchk(hipMemcpy(h_histogram.data(), d_histogram,
                         sizeof(size_t) * alphabet_size,
                         hipMemcpyDeviceToHost));

    for (size_t i = 0; i < alphabet_size; ++i) {
      EXPECT_EQ(hist_should[alphabet[i]], h_histogram[i]);
    }
    gpuErrchk(hipFree(d_alphabet));
    gpuErrchk(hipFree(d_histogram));
  }

  // Free memory
  gpuErrchk(hipFree(d_data));
}

TYPED_TEST(WaveletTreeTestFixture, structure) {
  std::vector<TypeParam> alphabet{0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  std::vector<TypeParam> data(100);
  for (size_t i = 0; i < data.size(); ++i) {
    data[i] = i % alphabet.size();
  }
  WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                            kGPUIndex);

  // First level
  for (size_t i = 0; i < data.size(); ++i) {
    // 0 to 7 is 0, 8 and 9 is 1
    BAaccessKernel<TypeParam>
        <<<1, 1>>>(wt.rank_select_.bit_array_, 0, i, this->result);
    kernelCheck();
    if (i % 10 < 8) {
      EXPECT_EQ(*this->result, 0);
    } else {
      EXPECT_EQ(*this->result, 1);
    }
  }

  // Second level
  for (size_t i = 0; i < 80; ++i) {
    // 4 0s and then 4 1s
    BAaccessKernel<TypeParam>
        <<<1, 1>>>(wt.rank_select_.bit_array_, 1, i, this->result);
    kernelCheck();
    if (i % 8 < 4) {
      EXPECT_EQ(*this->result, 0);
    } else {
      EXPECT_EQ(*this->result, 1);
    }
  }
  for (size_t i = 80; i < 100; ++i) {
    // 0101...
    BAaccessKernel<TypeParam>
        <<<1, 1>>>(wt.rank_select_.bit_array_, 1, i, this->result);
    kernelCheck();
    if (i % 2 == 0) {
      EXPECT_EQ(*this->result, 0);
    } else {
      EXPECT_EQ(*this->result, 1);
    }
  }

  // Third level
  for (size_t i = 0; i < 80; ++i) {
    // 2 0s and then 2 1s
    BAaccessKernel<TypeParam>
        <<<1, 1>>>(wt.rank_select_.bit_array_, 2, i, this->result);
    kernelCheck();
    if (i % 4 < 2) {
      EXPECT_EQ(*this->result, 0);
    } else {
      EXPECT_EQ(*this->result, 1);
    }
  }

  // Fourth level
  for (size_t i = 0; i < 80; ++i) {
    // 0101...
    BAaccessKernel<TypeParam>
        <<<1, 1>>>(wt.rank_select_.bit_array_, 3, i, this->result);
    kernelCheck();
    if (i % 2 == 0) {
      EXPECT_EQ(*this->result, 0);
    } else {
      EXPECT_EQ(*this->result, 1);
    }
  }
}

TYPED_TEST(WaveletTreeTestFixture, access) {
  std::vector<TypeParam> alphabet{0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  std::vector<TypeParam> data(100);
  for (size_t i = 0; i < data.size(); ++i) {
    data[i] = i % alphabet.size();
  }
  WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                            kGPUIndex);

  std::vector<size_t> indices(data.size());
  std::iota(indices.begin(), indices.end(), 0);
  compareAccessResults<TypeParam, 1>(wt, indices, data);
  compareAccessResults<TypeParam, 2>(wt, indices, data);
  compareAccessResults<TypeParam, 4>(wt, indices, data);
  compareAccessResults<TypeParam, 8>(wt, indices, data);
  compareAccessResults<TypeParam, 16>(wt, indices, data);
  compareAccessResults<TypeParam, 32>(wt, indices, data);
}

TYPED_TEST(WaveletTreeTestFixture, accessRandom) {
  for (int i = 0; i < 10; i++) {
    // Random data size between 1000 and 1'000'000
    size_t data_size = 1000 + (rand() % 1'000'000);
    size_t alphabet_size = std::min(
        data_size, size_t(rand() % std::numeric_limits<TypeParam>::max()));

    auto [alphabet, data] =
        generateRandomAlphabetAndData<TypeParam>(alphabet_size, data_size);
    alphabet_size = alphabet.size();

    WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                              kGPUIndex);

    // Create 100 random access queries
    std::vector<size_t> indices(100);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<size_t> dis(0, data_size - 1);
    std::generate(indices.begin(), indices.end(), [&]() { return dis(gen); });

    compareAccessResults<TypeParam, 1>(wt, indices, data);
    compareAccessResults<TypeParam, 2>(wt, indices, data);
    compareAccessResults<TypeParam, 4>(wt, indices, data);
    compareAccessResults<TypeParam, 8>(wt, indices, data);
    compareAccessResults<TypeParam, 16>(wt, indices, data);
    compareAccessResults<TypeParam, 32>(wt, indices, data);
  }
}

TYPED_TEST(WaveletTreeTestFixture, rank) {
  std::vector<TypeParam> alphabet{0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  std::vector<TypeParam> data(100);
  for (size_t i = 0; i < data.size(); ++i) {
    data[i] = i % alphabet.size();
  }
  WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                            kGPUIndex);

  std::vector<RankSelectQuery<TypeParam>> queries;
  for (size_t i = 0; i < data.size(); ++i) {
    queries.push_back({i, data[i]});
  }
  auto results = wt.rank(queries);
  for (size_t i = 0; i < data.size(); ++i) {
    EXPECT_EQ(i / 10, results[i]);
  }
}

TYPED_TEST(WaveletTreeTestFixture, rankRandom) {
  for (int i = 0; i < 10; i++) {
    // Random data size between 1000 and 1'000'000
    size_t data_size = 1000 + (rand() % 1'000'000);
    size_t alphabet_size =
        std::min(3 + (rand() % (data_size - 3)),
                 size_t(std::numeric_limits<TypeParam>::max()));

    auto [alphabet, data] =
        generateRandomAlphabetAndData<TypeParam>(alphabet_size, data_size);
    alphabet_size = alphabet.size();
    auto alphabet_copy = alphabet;

    WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                              kGPUIndex);

    // Create 100 random rank queries
    std::vector<RankSelectQuery<TypeParam>> queries(100);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<size_t> dis_index(0, data_size - 1);
    std::uniform_int_distribution<TypeParam> dis_alphabet(0, alphabet_size - 1);
    std::generate(queries.begin(), queries.end(), [&]() {
      return RankSelectQuery<TypeParam>{dis_index(gen),
                                        alphabet_copy[dis_alphabet(gen)]};
    });

    auto queries_copy = queries;
    auto results = wt.rank(queries_copy);
    for (size_t j = 0; j < queries.size(); ++j) {
      EXPECT_EQ(std::count(data.begin(), data.begin() + queries[j].index_,
                           queries[j].symbol_),
                results[j]);
    }
  }
}

TYPED_TEST(WaveletTreeTestFixture, select) {
  std::vector<TypeParam> alphabet{0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
  std::vector<TypeParam> data(100);
  for (size_t i = 0; i < data.size(); ++i) {
    data[i] = i % alphabet.size();
  }
  WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                            kGPUIndex);

  std::vector<RankSelectQuery<TypeParam>> queries;
  for (size_t i = 0; i < data.size(); ++i) {
    queries.push_back({i / 10 + 1, data[i]});
  }
  auto results = wt.select(queries);
  for (size_t i = 0; i < data.size(); ++i) {
    EXPECT_EQ(i, results[i]);
  }

  // Check that if there is no n-th occurrence of a symbol, the result is the
  // size of the data
  queries = std::vector<RankSelectQuery<TypeParam>>{{11, 0}};
  results = wt.select(queries);
  EXPECT_EQ(data.size(), results[0]);
}

TYPED_TEST(WaveletTreeTestFixture, selectRandom) {
  int num_iters = 10;
  int queries_per_iter = 100;
  if constexpr (std::is_same<TypeParam, uint64_t>::value or
                std::is_same<TypeParam, uint32_t>::value) {
    num_iters = 5;
    queries_per_iter = 10;
  }
  for (int i = 0; i < num_iters; i++) {
    // Random data size between 1000 and 1'000'000
    size_t data_size = 1000 + (rand() % 1'000'000);
    size_t alphabet_size =
        std::min(3 + (rand() % (data_size - 3)),
                 size_t(std::numeric_limits<TypeParam>::max()));

    std::unordered_map<TypeParam, size_t> hist;
    auto [alphabet, data] = generateRandomAlphabetDataAndHist<TypeParam>(
        alphabet_size, data_size, hist);
    alphabet_size = alphabet.size();
    auto alphabet_copy = alphabet;

    WaveletTree<TypeParam> wt(data.data(), data.size(), std::move(alphabet),
                              kGPUIndex);

    // Create 100 random select queries
    std::vector<RankSelectQuery<TypeParam>> queries(queries_per_iter);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<TypeParam> dis_alphabet(0, alphabet_size - 1);
    std::generate(queries.begin(), queries.end(), [&]() {
      TypeParam symbol_index;
      TypeParam symbol;
      size_t count;
      do {
        symbol_index = dis_alphabet(gen);
        symbol = alphabet_copy[symbol_index];
        count = hist[symbol];
      } while (count == 0);
      std::uniform_int_distribution<size_t> dis_index(1, count);
      auto index = dis_index(gen);

      return RankSelectQuery<TypeParam>{index, symbol};
    });

    auto queries_copy = queries;
    auto results = wt.select(queries_copy);
    for (size_t j = 0; j < queries.size(); ++j) {
      size_t counts = 0;
      EXPECT_EQ(std::find_if(data.begin(), data.end(),
                             [&](TypeParam c) {
                               return c == queries[j].symbol_ and
                                      ++counts == queries[j].index_;
                             }) -
                    data.begin(),
                results[j]);
    }
  }
}

/*
TYPED_TEST(WaveletTreeTestFixture, fillLevelRandom) {
  for (int i = 0; i < 100; i++) {
    // Random data size between 100 and 1'000'000
    size_t data_size = 100 + (rand() % 1'000'000);

    // Fill a vector with random data
    std::vector<TypeParam> data(data_size);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<TypeParam> dis(
        0, std::numeric_limits<TypeParam>::max());
    std::generate(data.begin(), data.end(), [&]() { return dis(gen); });

    // Copy the data to the device
    TypeParam* d_data;
    gpuErrchk(hipMalloc(&d_data, sizeof(TypeParam) * data_size));
    gpuErrchk(hipMemcpy(d_data, data.data(), sizeof(TypeParam) * data_size,
                         hipMemcpyHostToDevice));

    // CHoose a random alphabet start bit
    uint8_t num_bits = 8 * sizeof(TypeParam);
    uint8_t start_bit = rand() % num_bits;
    // Fill all levels from start_bit to 0
    std::vector<size_t> sizes(start_bit + 1, data_size);
    BitArray ba(sizes, false);
    for (uint8_t level = 0; level <= start_bit; ++level) {
      fillLevelKernel<TypeParam><<<1, 32>>>(ba, d_data, data_size, level);
      kernelCheck();

      // Check that the level is correctly filled
      std::vector<bool> level_should(data_size);
      for (size_t i = 0; i < data_size; ++i) {
        level_should[i] = getBit(start_bit - level, data[i]);
      }
    }
    hipFree(d_data);
  }
*/
}  // namespace ecl