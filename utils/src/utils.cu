#include "hip/hip_runtime.h"
#include <GPU_tunes.hpp>
#include <bit>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <limits>

#include "utils.cuh"

namespace ecl {
namespace internal {
static hipDeviceProp_t prop;
}  // namespace internal

__host__ std::pair<int, int> getLaunchConfig(size_t const num_warps,
                                             int const min_block_size,
                                             int max_block_size) {
  assert(internal::prop.totalGlobalMem != 0);
  int const min_block_size_warps = min_block_size / WS;
  int const warps_per_sm = internal::prop.maxThreadsPerMultiProcessor / WS;
  int const warps_per_block = internal::prop.maxThreadsPerBlock / WS;
  // find max block size that can still fully load an SM
  max_block_size = std::min(warps_per_block, max_block_size / WS);
  while (warps_per_sm % max_block_size != 0) {
    max_block_size -= 1;
  }
  if (num_warps <= max_block_size) {
    return {1, num_warps * WS};
  }
  std::pair<int, int> best_match = {-1, -1};
  int best_difference =
      std::numeric_limits<int>::max();  // Initialize with maximum value

  for (int k = 1; k <= max_block_size; ++k) {
    // Check if max_block_size is divisible by k
    if (max_block_size % k != 0) continue;

    // Calculate block_size and num_blocks
    int block_size = max_block_size / k;

    if (block_size < min_block_size_warps) {
      break;
    }
    int num_blocks_high = (num_warps + block_size - 1) / block_size;
    int num_blocks_low = num_warps / block_size;

    // Check if this is a perfect match
    if (num_warps % block_size == 0) {
      return {num_blocks_low, block_size * WS};
    }

    // Otherwise, calculate the difference and update best match if needed
    int difference = block_size * num_blocks_high - num_warps;
    if (difference < best_difference) {
      best_difference = difference;
      best_match = {num_blocks_high, block_size * WS};
    }
    difference = num_warps - block_size * num_blocks_low;
    if (difference < best_difference) {
      best_difference = difference;
      best_match = {num_blocks_low, block_size * WS};
    }
  }

  return best_match;  // Return the best match found
}

__host__ hipDeviceProp_t &getDeviceProperties() {
  assert(internal::prop.totalGlobalMem != 0);
  return internal::prop;
}

__host__ void checkWarpSize(uint8_t const GPU_index) {
  if (internal::prop.totalGlobalMem == 0) {
    hipGetDeviceProperties(&internal::prop, GPU_index);
    auto const threads_per_sm = internal::prop.maxThreadsPerMultiProcessor;
    kMaxTPB = internal::prop.maxThreadsPerBlock;
    // find max block size that can still fully load an SM
    while (threads_per_sm % kMaxTPB != 0) {
      kMaxTPB -= WS;
    }
    assert(kMaxTPB > WS);
    kMinBPM = threads_per_sm / kMaxTPB;
    auto const max_blocks_per_sm = internal::prop.maxBlocksPerMultiProcessor;
    kMinTPB = threads_per_sm / max_blocks_per_sm;
  }
  if (internal::prop.warpSize != WS) {
    fprintf(stderr, "Warp size must be 32, but is %d\n",
            internal::prop.warpSize);
    exit(EXIT_FAILURE);
  }
}

__host__ IdealConfigs &getIdealConfigs(const std::string &GPU_name) {
  auto get_configs = [](std::string GPU_name) {
    if (configs.find(GPU_name) != configs.end()) {
      return configs[GPU_name];
    } else {
      return IdealConfigs();
    }
  };
  static IdealConfigs ideal_configs = get_configs(GPU_name);
  return ideal_configs;
}
}  // namespace ecl