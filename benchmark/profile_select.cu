#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include <random>

#include "ecl_wavelet/tree/wavelet_tree.cuh"
#include "ecl_wavelet/utils/test_benchmark_utils.cuh"

template <typename T>
void profileSelect(size_t const data_size, size_t const alphabet_size,
                   size_t const num_queries, bool const use_profiler_api) {
  std::vector<T> alphabet;
  alphabet = std::vector<T>(alphabet_size);
  std::iota(alphabet.begin(), alphabet.end(),
            0);  // Min alphabet so that queries dont change

  auto [data, hist] = ecl::generateRandomDataAndHist<T>(alphabet, data_size);
  auto queries =
      ecl::generateRandomSelectQueries<T>(hist, num_queries, alphabet);
  ecl::WaveletTree<T> wt(data.data(), data_size, std::move(alphabet), 0);

  auto queries_copy = queries;
  auto results = wt.select(queries_copy.data(), num_queries);
  queries_copy = queries;
  if (use_profiler_api) {
    hipProfilerStart();
    results = wt.select(queries_copy.data(), num_queries);
    hipProfilerStop();
    queries_copy = queries;
  }
}

int main(int argc, char** argv) {
  // size is first command line argument
  auto const data_size = std::stoul(argv[1]);
  auto const alphabet_size = std::stoul(argv[2]);
  auto const num_queries = std::stoul(argv[3]);
  bool const use_profiler_api = argc > 4 ? std::stoi(argv[4]) : false;

  if (alphabet_size < std::numeric_limits<uint8_t>::max()) {
    profileSelect<uint8_t>(data_size, alphabet_size, num_queries,
                           use_profiler_api);
  } else if (alphabet_size < std::numeric_limits<uint16_t>::max()) {
    profileSelect<uint16_t>(data_size, alphabet_size, num_queries,
                            use_profiler_api);
  }
}