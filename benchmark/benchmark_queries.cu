#include "ecl_wavelet/tree/wavelet_tree.cuh"
#include "ecl_wavelet/utils/test_benchmark_utils.cuh"
#include "ecl_wavelet/utils/utils.cuh"

namespace ecl {

template <typename T>
size_t processAccessQueries(WaveletTree<T>& wt, std::vector<size_t>& queries,
                            size_t const num_iters) {
  std::vector<size_t> times(num_iters);
  auto start = std::chrono::high_resolution_clock::now();
  auto end = std::chrono::high_resolution_clock::now();

  // warm-up
  for (size_t i = 0; i < 2; ++i) {
    auto results = wt.access(queries.data(), queries.size());
  }
  for (size_t i = 0; i < num_iters; ++i) {
    start = std::chrono::high_resolution_clock::now();
    auto results = wt.access(queries.data(), queries.size());
    end = std::chrono::high_resolution_clock::now();
    times[i] =
        std::chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count();
  }

  std::nth_element(times.begin(), times.begin() + times.size() / 2,
                   times.end());
  return times[times.size() / 2];
}
template <typename T, bool DoRank>
size_t processRSQueries(WaveletTree<T>& wt,
                        std::vector<RankSelectQuery<T>>& queries,
                        size_t const num_iters) {
  std::vector<size_t> times(num_iters);
  auto start = std::chrono::high_resolution_clock::now();
  auto end = std::chrono::high_resolution_clock::now();

  // warm-up
  for (size_t i = 0; i < 2; ++i) {
    if constexpr (DoRank) {
      auto results = wt.rank(queries.data(), queries.size());
    } else {
      auto results = wt.select(queries.data(), queries.size());
    }
  }
  for (size_t i = 0; i < num_iters; ++i) {
    start = std::chrono::high_resolution_clock::now();
    if constexpr (DoRank) {
      auto results = wt.rank(queries.data(), queries.size());
    } else {
      auto results = wt.select(queries.data(), queries.size());
    }
    end = std::chrono::high_resolution_clock::now();
    times[i] =
        std::chrono::duration_cast<std::chrono::microseconds>(end - start)
            .count();
  }

  std::nth_element(times.begin(), times.begin() + times.size() / 2,
                   times.end());
  return times[times.size() / 2];
}

template <typename T>
static void BM_queries(size_t const data_size, size_t const alphabet_size,
                       std::vector<size_t> const& num_queries_vec,
                       size_t const num_iters, uint32_t const GPU_index,
                       std::string const& output) {
  std::vector<T> alphabet(alphabet_size);
  std::iota(alphabet.begin(), alphabet.end(), 0ULL);
  auto const [data, hist] = generateRandomDataAndHist(alphabet, data_size);

  auto alphabet_copy = alphabet;
  WaveletTree<T> wt(data.data(), data_size, std::move(alphabet_copy),
                    GPU_index);

  for (auto const num_queries : num_queries_vec) {
    auto access_queries = generateRandomAccessQueries(data_size, num_queries);
    auto rank_queries =
        generateRandomRankQueries(data_size, num_queries, alphabet);
    auto select_queries =
        generateRandomSelectQueries(hist, num_queries, alphabet);

    size_t const median_access_time =
        processAccessQueries(wt, access_queries, num_iters);
    gpuErrchk(hipHostRegister(access_queries.data(),
                               access_queries.size() * sizeof(size_t),
                               hipHostRegisterPortable));
    size_t const median_access_time_pinned =
        processAccessQueries(wt, access_queries, num_iters);
    gpuErrchk(hipHostUnregister(access_queries.data()));

    size_t const median_rank_time =
        processRSQueries<T, true>(wt, rank_queries, num_iters);
    size_t const median_select_time =
        processRSQueries<T, false>(wt, select_queries, num_iters);

    gpuErrchk(hipHostRegister(rank_queries.data(),
                               rank_queries.size() * sizeof(RankSelectQuery<T>),
                               hipHostRegisterPortable));
    gpuErrchk(
        hipHostRegister(select_queries.data(),
                         select_queries.size() * sizeof(RankSelectQuery<T>),
                         hipHostRegisterPortable));
    size_t const median_rank_time_pinned =
        processRSQueries<T, true>(wt, rank_queries, num_iters);
    size_t const median_select_time_pinned =
        processRSQueries<T, false>(wt, select_queries, num_iters);

    std::sort(
        rank_queries.begin(), rank_queries.end(),
        [](auto const& a, auto const& b) { return a.symbol_ < b.symbol_; });
    std::sort(
        select_queries.begin(), select_queries.end(),
        [](auto const& a, auto const& b) { return a.symbol_ < b.symbol_; });

    size_t const median_rank_time_sorted_pinned =
        processRSQueries<T, true>(wt, rank_queries, num_iters);
    size_t const median_select_time_sorted_pinned =
        processRSQueries<T, false>(wt, select_queries, num_iters);

    gpuErrchk(hipHostUnregister(rank_queries.data()));
    gpuErrchk(hipHostUnregister(select_queries.data()));

    size_t const median_rank_time_sorted =
        processRSQueries<T, true>(wt, rank_queries, num_iters);
    size_t const median_select_time_sorted =
        processRSQueries<T, false>(wt, select_queries, num_iters);
    std::ofstream file(output, std::ios_base::app);
    file << data_size << "," << alphabet_size << "," << num_queries << ","
         << median_access_time << "," << median_access_time_pinned << ","
         << median_rank_time << "," << median_select_time << ","
         << median_rank_time_sorted << "," << median_select_time_sorted << ","
         << median_rank_time_pinned << "," << median_select_time_pinned << ","
         << median_rank_time_sorted_pinned << ","
         << median_select_time_sorted_pinned << std::endl;
    file.close();
  }
}

}  // namespace ecl

int main(int argc, char** argv) {
  if (argc != 3) {
    std::cerr << "Usage: " << argv[0] << "<num_iters> <output_dir> <GPU_index>"
              << std::endl;
    return EXIT_FAILURE;
  }

  uint32_t const num_iters = std::stoi(argv[1]);
  std::string const output_dir = argv[2];
  uint32_t const GPU_index = std::stoi(argv[3]);

  size_t const data_size = 6'000'000'000;

  std::vector<size_t> const num_queries_vec{100'000,   500'000,    1'000'000,
                                            5'000'000, 10'000'000, 100'000'000};

  std::vector<size_t> const alphabet_sizes{
      4,    6,    8,    12,    16,    24,    32,    48,    64,   96,
      128,  192,  256,  384,   512,   768,   1024,  1536,  2048, 3072,
      4096, 6144, 8192, 12288, 16384, 24576, 32768, 49152, 65536};

  auto out_file = output_dir + "/ecl_queries_random_bm.csv";
  std::ofstream file(out_file);
  file << "data_size,alphabet_size,num_queries,median_access_time(mus),median_"
          "access_time_pinned,"
          "median_rank_time ,median_select_time,"
          "median_rank_time_sorted, "
          "median_select_time_sorted,median_rank_time_pinned,"
          "median_select_time_pinned,median_rank_time_sorted_pinned,"
          "median_select_time_sorted_pinned"
       << std::endl;
  file.close();

  for (auto const alphabet_size : alphabet_sizes) {
    if (alphabet_size <= 256) {
      ecl::BM_queries<uint8_t>(data_size, alphabet_size, num_queries_vec,
                               num_iters, GPU_index, out_file);
    } else {
      ecl::BM_queries<uint16_t>(data_size / 2, alphabet_size, num_queries_vec,
                                num_iters, GPU_index, out_file);
    }
  }
  return EXIT_SUCCESS;
}