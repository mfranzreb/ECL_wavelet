#include "hip/hip_runtime.h"
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/remove.h>

#include <algorithm>
#include <cmath>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_scan.cuh>
#include <numeric>
#include <vector>

#include "rank_select.cuh"
#include "utils.cuh"
#include "wavelet_tree.cuh"

namespace ecl {
__device__ size_t d_data_len;
typedef unsigned long long int cu_size_t;

// TODO: How to circumvent this
template class WaveletTree<uint8_t>;
template class WaveletTree<uint16_t>;
template class WaveletTree<uint32_t>;
template class WaveletTree<uint64_t>;

template <typename T>
__host__ WaveletTree<T>::WaveletTree(T* const data, size_t data_size,
                                     std::vector<T>&& alphabet)
    : alphabet_(alphabet), is_copy_(false) {
  static_assert(std::is_integral<T>::value and std::is_unsigned<T>::value,
                "T must be an unsigned integral type");
  assert(data_size > 0);
  assert(alphabet.size() > 0);
  assert(std::is_sorted(alphabet.begin(), alphabet.end()));
  checkWarpSize();
  // make minimal alphabet
  alphabet_size_ = alphabet.size();
  std::vector<T> min_alphabet(alphabet_size_);
  std::iota(min_alphabet.begin(), min_alphabet.end(), 0);

  num_levels_ = ceil(log2(alphabet_size_));
  alphabet_start_bit_ = num_levels_ - 1;

  // copy minimal alphabet to device
  gpuErrchk(hipMalloc(&d_min_alphabet_, alphabet_size_ * sizeof(T)));
  gpuErrchk(hipMemcpy(d_min_alphabet_, min_alphabet.data(),
                       alphabet_size_ * sizeof(T), hipMemcpyHostToDevice));

  // create codes and copy to device
  std::vector<Code> codes = createMinimalCodes(min_alphabet);
  gpuErrchk(hipMalloc(&d_codes_, alphabet_size_ * sizeof(Code)));
  gpuErrchk(hipMemcpy(d_codes_, codes.data(), alphabet_size_ * sizeof(Code),
                       hipMemcpyHostToDevice));

  std::vector<uint8_t> code_lens(codes.back().code_ + 1);
  for (size_t i = 0; i < alphabet_size_; ++i) {
    code_lens[codes[i].code_] = codes[i].len_;
  }
  gpuErrchk(hipMalloc(&d_code_lens_, code_lens.size() * sizeof(uint8_t)));
  gpuErrchk(hipMemcpy(d_code_lens_, code_lens.data(),
                       code_lens.size() * sizeof(uint8_t),
                       hipMemcpyHostToDevice));

  // Allocate space for counts array
  gpuErrchk(hipMalloc(&d_counts_, alphabet_size_ * sizeof(size_t)));
  gpuErrchk(hipMemset(d_counts_, 0, alphabet_size_ * sizeof(size_t)));

  // Copy data to device
  T* d_data;
  gpuErrchk(hipMalloc(&d_data, data_size * sizeof(T)));
  gpuErrchk(
      hipMemcpy(d_data, data, data_size * sizeof(T), hipMemcpyHostToDevice));

  // Copy alphabet to device
  T* d_alphabet;
  gpuErrchk(hipMalloc(&d_alphabet, alphabet_size_ * sizeof(T)));
  gpuErrchk(hipMemcpy(d_alphabet, alphabet.data(), alphabet_size_ * sizeof(T),
                       hipMemcpyHostToDevice));

  // Compute global_histogram and change text to min_alphabet
  // TODO: find appropriate number of warps
  auto num_warps = std::min((data_size + WS - 1) / WS, 1'000'000UL);
  auto [num_blocks, threads_per_block] = getLaunchConfig(num_warps, 32, 1024);
  computeGlobalHistogramKernel<T><<<num_blocks, threads_per_block>>>(
      *this, d_data, data_size, d_counts_, d_alphabet, alphabet_size_);

  // Copy counts to host
  std::vector<size_t> counts(alphabet_size_);
  gpuErrchk(hipMemcpy(counts.data(), d_counts_,
                       alphabet_size_ * sizeof(size_t),
                       hipMemcpyDeviceToHost));

  // Calculate size of bit array at each level
  std::vector<size_t> bit_array_sizes(num_levels_, data_size);
  // Get min code length
  uint8_t min_code_len = codes.back().len_;
  for (size_t i = num_levels_ - 1; i >= min_code_len; --i) {
    for (int64_t j = alphabet_size_ - 1; j >= 0; --j) {
      if (i >= codes[j].len_) {
        bit_array_sizes[i] -= counts[j];
      } else {
        break;
      }
    }
  }

  // Perform exclusive sum of counts
  void* d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(nullptr, temp_storage_bytes, d_counts_,
                                d_counts_, alphabet_size_);

  gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
  hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_counts_,
                                d_counts_, alphabet_size_);
  gpuErrchk(hipFree(d_temp_storage));

  BitArray bit_array(bit_array_sizes, false);

  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_data_len), &data_size, sizeof(size_t),
                               size_t(0), hipMemcpyHostToDevice));
  fillLevelKernel<T><<<num_blocks, threads_per_block>>>(bit_array, d_data,
                                                        alphabet_start_bit_, 0);

  // Allocate space for sorted data
  T* d_sorted_data;
  gpuErrchk(hipMalloc(&d_sorted_data, data_size * sizeof(T)));

  d_temp_storage = nullptr;
  temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data,
                                 d_sorted_data, data_size);
  gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));

  data_size = bit_array_sizes[1];
  for (uint32_t l = 1; l < num_levels_; l++) {
    assert(data_size == bit_array_sizes[l]);
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_data_len), &data_size, sizeof(size_t),
                                 size_t(0), hipMemcpyHostToDevice));
    // Perform radix sort
    hipcub::DeviceRadixSort::SortKeys(
        d_temp_storage, temp_storage_bytes, d_data, d_sorted_data, data_size,
        alphabet_start_bit_ + 1 - l, alphabet_start_bit_ + 1);
    // TODO, could launch in different streams
    //  Fill l-th bit array
    fillLevelKernel<T><<<num_blocks, threads_per_block>>>(
        bit_array, d_sorted_data, alphabet_start_bit_, l);
    kernelCheck();

    if (l != (num_levels_ - 1) and
        bit_array_sizes[l] != bit_array_sizes[l + 1]) {
      //  Reduce text
      T* new_end = thrust::remove_if(thrust::device, d_data, d_data + data_size,
                                     isLongEnough<T>(d_code_lens_, l));
      data_size = static_cast<size_t>(std::distance(d_data, new_end));
    }
  }

  gpuErrchk(hipFree(d_data));
  gpuErrchk(hipFree(d_sorted_data));
  gpuErrchk(hipFree(d_alphabet));
  gpuErrchk(hipFree(d_temp_storage));

  // build rank and select structures from bit-vectors
  rank_select_ = RankSelect(std::move(bit_array));
}

template <typename T>
__host__ WaveletTree<T>::WaveletTree(WaveletTree const& other)
    : alphabet_(other.alphabet_),
      rank_select_(other.rank_select_),
      alphabet_size_(other.alphabet_size_),
      alphabet_start_bit_(other.alphabet_start_bit_),
      num_levels_(other.num_levels_),
      d_min_alphabet_(other.d_min_alphabet_),
      d_codes_(other.d_codes_),
      d_code_lens_(other.d_code_lens_),
      d_counts_(other.d_counts_),
      is_copy_(true) {}

template <typename T>
WaveletTree<T>::~WaveletTree() {
  if (not is_copy_) {
    gpuErrchk(hipFree(d_min_alphabet_));
    gpuErrchk(hipFree(d_codes_));
    gpuErrchk(hipFree(d_code_lens_));
    gpuErrchk(hipFree(d_counts_));
  }
}

template <typename T>
__host__ std::vector<T> WaveletTree<T>::access(
    std::vector<size_t> const& indices) {
  // launch kernel with 1 warp per index
  size_t num_indices = indices.size();
  auto [num_blocks, threads_per_block] = getLaunchConfig(num_indices, 32, 1024);

  // allocate space for results
  T* d_results;
  gpuErrchk(hipMalloc(&d_results, num_indices * sizeof(T)));

  // Copy indices to device
  size_t* d_indices;
  gpuErrchk(hipMalloc(&d_indices, num_indices * sizeof(size_t)));
  gpuErrchk(hipMemcpy(d_indices, indices.data(), num_indices * sizeof(size_t),
                       hipMemcpyHostToDevice));

  accessKernel<T><<<num_blocks, threads_per_block>>>(*this, d_indices,
                                                     num_indices, d_results);
  kernelCheck();

  // copy results back to host
  std::vector<T> results(num_indices);
  gpuErrchk(hipMemcpy(results.data(), d_results, num_indices * sizeof(T),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(d_indices));
  gpuErrchk(hipFree(d_results));

#pragma omp parallel for
  for (size_t i = 0; i < num_indices; ++i) {
    results[i] = alphabet_[results[i]];
  }
  return results;
}

template <typename T>
__host__ std::vector<size_t> WaveletTree<T>::rank(
    std::vector<RankSelectQuery<T>>& queries) {
  // launch kernel with 1 warp per index
  size_t const num_queries = queries.size();
  auto [num_blocks, threads_per_block] = getLaunchConfig(num_queries, 32, 1024);

  // allocate space for results
  size_t* d_results;
  gpuErrchk(hipMalloc(&d_results, num_queries * sizeof(size_t)));

  // Convert query symbols to minimal alphabet
#pragma omp parallel for
  for (size_t i = 0; i < num_queries; ++i) {
    auto const symbol_index =
        std::lower_bound(alphabet_.begin(), alphabet_.end(),
                         queries[i].symbol_) -
        alphabet_.begin();
    assert(symbol_index < alphabet_size_);
    queries[i].symbol_ = static_cast<T>(symbol_index);
  }

  // Copy queries to device
  RankSelectQuery<T>* d_queries;
  gpuErrchk(hipMalloc(&d_queries, num_queries * sizeof(RankSelectQuery<T>)));
  gpuErrchk(hipMemcpy(d_queries, queries.data(),
                       num_queries * sizeof(RankSelectQuery<T>),
                       hipMemcpyHostToDevice));

  rankKernel<T><<<num_blocks, threads_per_block>>>(*this, d_queries,
                                                   num_queries, d_results);
  kernelCheck();

  // copy results back to host
  std::vector<size_t> results(num_queries);
  gpuErrchk(hipMemcpy(results.data(), d_results, num_queries * sizeof(size_t),
                       hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(d_queries));
  gpuErrchk(hipFree(d_results));

  return results;
}

template <typename T>
__host__ std::vector<size_t> WaveletTree<T>::select(
    std::vector<RankSelectQuery<T>> const& queries) {
  // 1 warp per query
  // group characters together to reduce memory access
  return std::vector<size_t>(1);
}

template <typename T>
__device__ WaveletTree<T>::Code WaveletTree<T>::encode(T const c) {
  assert(c < alphabet_size_);
  return d_codes_[c];
}

template <typename T>
__host__ std::vector<typename WaveletTree<T>::Code>
WaveletTree<T>::createMinimalCodes(std::vector<T> const& alphabet) {
  auto const alphabet_size = alphabet.size();
  std::vector<Code> codes(alphabet_size);
  uint8_t const total_num_bits = ceil(log2(alphabet_size));
  uint8_t const alphabet_start_bit = total_num_bits - 1;
#pragma omp parallel for
  for (size_t i = 0; i < alphabet_size; ++i) {
    codes[i].len_ = total_num_bits;
    codes[i].code_ = i;
  }
  if (isPowTwo<size_t>(alphabet_size)) {
    return codes;
  }
  uint8_t start_bit = 0;  // 0 is the alphabet start bit.
  size_t start_i = 0;
  uint8_t code_len = total_num_bits;
  size_t num_codes = alphabet_size;
  do {
    for (uint32_t i = code_len - 1; i > 0; --i) {
      auto pow_two = powTwo<uint32_t>(i);
      if (num_codes <= pow_two) {
        break;
      }
      num_codes -= pow_two;
      start_i += pow_two;
      start_bit++;
    }
    if (num_codes == 1) {
      code_len = 1;
      codes[alphabet_size - 1].len_ = start_bit;
      codes[alphabet_size - 1].code_ = ((1UL << start_bit) - 1)
                                       << (alphabet_start_bit + 1 - start_bit);
    } else {
      code_len = ceil(log2(num_codes));
#pragma omp parallel for
      for (int i = alphabet_size - num_codes; i < alphabet_size; i++) {
        // Code of local subtree
        T code = i - start_i;
        // Shift code to start at start_bit
        code <<= (total_num_bits - start_bit - code_len);
        // Add to global code already saved
        code += (~((1UL << (total_num_bits - start_bit)) - 1)) & codes[i].code_;

        codes[i].code_ = code;
        codes[i].len_ = start_bit + code_len;
      }
    }
  } while (code_len > 1);
  return codes;
}

template <typename T>
__device__ size_t WaveletTree<T>::getAlphabetSize() const {
  return alphabet_size_;
}

template <typename T>
__device__ size_t WaveletTree<T>::getNumLevels() const {
  return num_levels_;
}

template <typename T>
__device__ size_t WaveletTree<T>::getCounts(size_t i) const {
  return d_counts_[i];
}

template <typename T>
__global__ void computeGlobalHistogramKernel(WaveletTree<T> tree, T* data,
                                             size_t const data_size,
                                             size_t* counts, T* const alphabet,
                                             size_t const alphabet_size) {
  assert(blockDim.x % WS == 0);
  uint32_t total_threads = blockDim.x * gridDim.x;
  uint32_t global_t_id = blockIdx.x * blockDim.x + threadIdx.x;
  for (uint32_t i = global_t_id; i < data_size; i += total_threads) {
    T const char_data = data[i];
    size_t const char_index =
        thrust::lower_bound(thrust::seq, alphabet, alphabet + alphabet_size,
                            char_data) -
        alphabet;
    typename WaveletTree<T>::Code const code = tree.encode(char_index);
    atomicAdd((cu_size_t*)&counts[char_index], size_t(1));
    data[i] = code.code_;
  }
}

template <typename T>
__global__ void fillLevelKernel(BitArray bit_array, T* const data,
                                uint8_t const alphabet_start_bit,
                                uint32_t const level) {
  assert(blockDim.x % WS == 0);
  uint32_t const global_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / WS;
  uint32_t const num_warps = gridDim.x * blockDim.x / WS;
  uint8_t const local_t_id = threadIdx.x % WS;

  size_t const start = WS * global_warp_id;
  // Each warp processes a block of data
  for (uint32_t i = start; i < d_data_len; i += WS * num_warps) {
    T code;
    if (i + local_t_id >= d_data_len) {
      code = 0;
    } else {
      code = data[i + local_t_id];
    }

    // Warp vote to all the bits that need to get written to a word
    uint32_t word = __ballot_sync(~0, getBit(alphabet_start_bit - level, code));

    if (local_t_id == 0) {
      bit_array.writeWordAtBit(level, i, word);
    }
  }
}

template <typename T>
__global__ void accessKernel(WaveletTree<T> tree, size_t* const indices,
                             size_t const num_indices, T* results) {
  assert(blockDim.x % WS == 0);
  uint32_t const global_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / WS;
  uint32_t const num_warps = gridDim.x * blockDim.x / WS;
  uint32_t const local_t_id = threadIdx.x % WS;

  for (uint32_t i = global_warp_id; i < num_indices; i += num_warps) {
    size_t index = indices[i];

    uint32_t char_start = 0;
    uint32_t char_end = tree.getAlphabetSize();
    uint32_t start, pos;
    for (uint32_t l = 0; l < tree.getNumLevels(); ++l) {
      if (char_end - char_start == 1) {
        break;
      }
      // TODO: could be done in parallel, and combined if index is less than L2
      // block size
      start = tree.rank_select_.rank0(l, tree.getCounts(char_start), local_t_id,
                                      WS);
      pos = tree.rank_select_.rank0(l, tree.getCounts(char_start) + index,
                                    local_t_id, WS);
      if (tree.rank_select_.bit_array_.access(
              l, tree.getCounts(char_start) + index) == false) {
        index = pos - start;
        char_end = char_start + getPrevPowTwo(char_end - char_start);
      } else {
        index -= pos - start;
        char_start += getPrevPowTwo(char_end - char_start);
      }
    }
    results[i] = char_start;
  }
}

template <typename T>
__global__ void rankKernel(WaveletTree<T> tree,
                           RankSelectQuery<T>* const queries,
                           size_t const num_queries, size_t* const ranks) {
  assert(blockDim.x % WS == 0);
  uint32_t const global_warp_id = (blockIdx.x * blockDim.x + threadIdx.x) / WS;
  uint32_t const num_warps = gridDim.x * blockDim.x / WS;
  uint32_t const local_t_id = threadIdx.x % WS;

  for (uint32_t i = global_warp_id; i < num_queries; i += num_warps) {
    RankSelectQuery<T> query = queries[i];

    uint32_t char_start = 0;
    uint32_t char_end = tree.getAlphabetSize();
    uint32_t char_split;
    uint32_t start, pos;
    for (uint32_t l = 0; l < tree.getNumLevels(); ++l) {
      if (char_end - char_start == 1) {
        break;
      }
      // TODO: could be done in parallel, and combined if index is less than L2
      // block size
      start = tree.rank_select_.rank0(l, tree.getCounts(char_start), local_t_id,
                                      WS);
      pos = tree.rank_select_.rank0(
          l, tree.getCounts(char_start) + query.index_, local_t_id, WS);
      char_split = char_start + getPrevPowTwo(char_end - char_start);
      if (query.symbol_ < char_split) {
        query.index_ = pos - start;
        char_end = char_split;
      } else {
        query.index_ -= pos - start;
        char_start = char_split;
      }
    }
    ranks[i] = query.index_;
  }
}

template <typename T>
__global__ void selectKernel(WaveletTree<T> tree,
                             RankSelectQuery<T>* const queries,
                             size_t const num_queries, size_t* const ranks) {
  // for l = to to code length
}
}  // namespace ecl